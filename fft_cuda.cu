#include "hip/hip_runtime.h"
#include "fft_cuda.h"
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <cstdio>

__device__ __host__ cpxcuda operator+ (const cpxcuda &a, const cpxcuda &b) {
    cpxcuda result;
    result.re = a.re + b.re;
    result.im = a.im + b.im;
    return result;
}

__device__ __host__ cpxcuda operator- (const cpxcuda &a, const cpxcuda &b) {
    cpxcuda result;
    result.re = a.re - b.re;
    result.im = a.im - b.im;
    return result;
}

__device__ __host__ cpxcuda operator* (const cpxcuda &a, const cpxcuda &b) {
    cpxcuda result;
    result.re = a.re * b.re - a.im * b.im;
    result.im = a.re * b.im + a.im * b.re;
    return result;
}

cpxcuda e_imaginary(double x) {
    cpxcuda w;
    w.re = cos(x);
    w.im = sin(x);
    return w;
}

__global__ void kernel(cpxcuda *out_device, cpxcuda *ws, int n, int batch, bool reverse) {
    int idX = blockIdx.x;
    int idY = threadIdx.y;
    for (len_t block_size = 2, p = 0; block_size <= n; block_size *= 2, p++) {
        len_t half_block_size = block_size >> 1;
        int step = n / block_size;
        for (len_t i = idY; i < n/2; i += blockDim.y) {
            len_t j = i & (half_block_size - 1);
            len_t index = ((i >> p) << (p + 1)) + j;
            cpxcuda x = out_device[idX * n + index];
            cpxcuda y = out_device[idX * n + (index + half_block_size)] * ws[j * step];
            out_device[idX * n + index] = x + y;
            out_device[idX * n + (index + half_block_size)] = x - y;
        }
        __syncthreads();
    }
    if (reverse) {
        cpxcuda mult;
        mult.re = 1/(double)n;
        mult.im = 0;
        for (len_t i = idY; i < n; i += blockDim.y) {
            out_device[idX * n + i] = out_device[idX * n + i] * mult;
        }
    }
}

fft_plan_cuda fft_plan_cuda_1d(int n, int batch, cpxcuda *in, cpxcuda *out, bool reverse) {
    int upper_n = 1;
    while (upper_n < n) upper_n <<= 1;
    
    // swap
    int *index_mapping = (int*) calloc(upper_n, sizeof(int));
    int bit=0;
    while ((1<<bit) < upper_n) bit++;
    for (int i = 0; i < upper_n - 1; i++) {
        index_mapping[i] = (index_mapping[i >> 1] >> 1) | ((i & 1) << (bit - 1));
    }
    index_mapping[upper_n - 1] = upper_n - 1;
    for (int i = 0; i < batch; i++)
        for (int j = 0; j < upper_n; j++)
            out[i * upper_n + index_mapping[j]] = in[i * upper_n + j];

    cpxcuda *out_device = NULL;
    hipMalloc(&out_device, sizeof(cpxcuda) * batch * upper_n);
    hipMemcpy(out_device, out, sizeof(cpxcuda) * upper_n * batch, hipMemcpyHostToDevice);

    cpxcuda *ws = (cpxcuda *) calloc(upper_n / 2, sizeof(cpxcuda));
    ws[0].re = 1; ws[0].im = 0;
    ws[1] = e_imaginary(double(2 * M_PI) / double(upper_n) * (reverse ? -1 : 1)); 
    for (int i = 2; i < upper_n / 2; i++)
        ws[i] = ws[i - 1] * ws[1];

    cpxcuda *ws_device = NULL;
    hipMalloc(&ws_device, sizeof(cpxcuda) * upper_n / 2);
    hipMemcpy(ws_device, ws, sizeof(cpxcuda) * upper_n / 2, hipMemcpyHostToDevice);

    free(ws);
    free(index_mapping);
    
    fft_plan_cuda plan;
    plan.n = upper_n;
    plan.batch = batch;
    plan.out = out;
    plan.out_device = out_device;
    plan.ws_device = ws_device;
    plan.reverse = reverse;

    return plan;
}

void fft_execute_plan_cuda(fft_plan_cuda &plan) {
    dim3 blockDim(1, 256);
    dim3 gridDim(plan.batch, 1);
    kernel<<<gridDim, blockDim>>>(plan.out_device, plan.ws_device, plan.n, plan.batch, plan.reverse);
    hipDeviceSynchronize();
    hipMemcpy(plan.out, plan.out_device, sizeof(cpxcuda) * plan.n * plan.batch, hipMemcpyDeviceToHost);
}

void fft_destroy_plan_cuda(fft_plan_cuda &plan) {
    hipFree(plan.ws_device);
    hipFree(plan.out_device);
}

//reference: CUDA example slides.